
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <memory>
#include <iostream>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    exit(EXIT_FAILURE);}} while(0)

using namespace std;

template<typename T>
class device_copyable
{
public:
	virtual void copyToDevice(T** pos) = 0;
	virtual ~device_copyable() = default;
};

class Base
{
public:
	int b_id;
public:
	__host__ __device__ Base(int _b_id):b_id(_b_id){}
	__host__ __device__ Base(const Base& b):b_id(b.b_id){}
	virtual ~Base() = default;
	__device__ virtual void f(){};
};

class Sub1 : public Base
{
public:
	int sub_id;
public:
	__host__ __device__ Sub1(int _b_id, int _sub_id):Base(_b_id), sub_id(_sub_id){}
	__host__ __device__ Sub1(const Sub1& s):Base(s), sub_id(s.sub_id){}
	__device__ void f() override
	{
		printf("hello 1: %d %d!\n", b_id, sub_id);
	}
};

class Sub1Host : public Sub1, public device_copyable<Base>
{
public:
	using Sub1::Sub1;
	void copyToDevice(Base** pos) override;
};

__global__ void create_sub1(Base** object, Sub1Host h)
{
	*object = new Sub1(h);
}

void Sub1Host::copyToDevice(Base** pos)
{
	create_sub1<<<1,1>>>(pos, *this);
}

__global__ void runner(Base** objects, int len)
{
	for (int i=0; i<len; i++)
		objects[i]->f();
}

void run(std::vector<std::unique_ptr<device_copyable<Base>>> objs)
{
	Base** d_objects;
	CUDA_CALL(hipMalloc((void **)&d_objects, objs.size() * sizeof(Base*)));
	for (int i=0; i<objs.size(); i++)
	{
		objs[i]->copyToDevice(&d_objects[i]);
	}
	runner<<<1,1>>>(d_objects, objs.size());
}

int main()
{
	std::vector<std::unique_ptr<device_copyable<Base>>> objs;
	objs.push_back(std::unique_ptr<device_copyable<Base>>(new Sub1Host(42, 68)));
	run(std::move(objs));
	hipDeviceSynchronize();
	return 0;
}
