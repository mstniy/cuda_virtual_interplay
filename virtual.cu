
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <memory>
#include <iostream>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    exit(EXIT_FAILURE);}} while(0)

using namespace std;

class Base
{
public:
	int b_id;
public:
	Base() = default;
	__host__ __device__ Base(int _b_id):b_id(_b_id){}
	virtual ~Base() = default;
};

class BaseDevice;

class BaseHost : public virtual Base
{
public:
	virtual void createOnDevice(BaseDevice** pos) = 0;
};

class BaseDevice : public virtual Base
{
public:
	__device__ virtual void f() = 0;
};

class Sub1 : public virtual Base
{
public:
	int sub_id;
public:
	__host__ __device__ Sub1(int _sub_id):sub_id(_sub_id){}
};

class Sub1Host : public BaseHost, public Sub1
{
public:
	Sub1Host(int _b_id, int _sub_id):Base(_b_id), Sub1(_sub_id){}
	void createOnDevice(BaseDevice** pos) override;
};

class Sub1Device : public BaseDevice, public Sub1
{
public:
	__device__ Sub1Device(int _b_id, int _sub_id):Base(_b_id), Sub1(_sub_id){}
public:
	__device__ void f() override
	{
		printf("hello 1: %d %d!\n", b_id, sub_id);
	}
};

__global__ void create_sub1(BaseDevice** object, int base_id, int sub_id)
{
	*object = new Sub1Device(base_id, sub_id);
}

void Sub1Host::createOnDevice(BaseDevice** pos)
{
	create_sub1<<<1,1>>>(pos, b_id, sub_id);
}

__global__ void runner(BaseDevice** objects, int len)
{
	for (int i=0; i<len; i++)
		objects[i]->f();
}

void run(std::vector<std::unique_ptr<BaseHost>> objs)
{
	BaseDevice** d_objects;
	CUDA_CALL(hipMalloc((void **)&d_objects, objs.size() * sizeof(BaseDevice*)));
	for (int i=0; i<objs.size(); i++)
	{
		objs[i]->createOnDevice(&d_objects[i]);
	}
	runner<<<1,1>>>(d_objects, objs.size());
}

int main()
{
	std::vector<std::unique_ptr<BaseHost>> funcs;
	funcs.push_back(std::unique_ptr<BaseHost>(new Sub1Host(42, 68)));
	run(std::move(funcs));
	hipDeviceSynchronize();
	return 0;
}
