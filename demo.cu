#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <memory>
#include <iostream>
#include "virtual_interplay.h"

#ifndef CUDA_CALL
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    exit(EXIT_FAILURE);}} while(0)
#endif

using namespace std;

class Base : public Managed
{
public:
	int b_id;
public:
	Base() = default;
	__host__ __device__ Base(int _b_id):b_id(_b_id){}
	virtual ~Base() = default;
	__device__ virtual void f() = 0;
	__host__ __device__ virtual void g() = 0;
};

// Classes with virtual bases are supported
class Sub1 : virtual public Base
{
public:
	int sub_id;
public:
	Sub1() = default;
	__host__ __device__ Sub1(int _b_id, int _sub_id):Base(_b_id), sub_id(_sub_id){}
	__device__ void f() override
	{
		printf("Sub1::f %d %d!\n", b_id, sub_id);
		b_id++;
		sub_id++;
	}
	__host__ __device__ void g() override
	{
		printf("Sub1::g %d %d!\n", b_id, sub_id);
	}
};

class Sub2 : public Base
{
public:
	char sub_ch;
public:
	Sub2() = delete; // non-default constructible classes are also supported
	__host__ __device__ Sub2(int _b_id, char _sub_ch):Base(_b_id), sub_ch(_sub_ch){}
	Sub2(const Sub2&) = delete; // non-copy constructible classes are also supported
	Sub2(Sub2&& s) = default;
	__device__ void f() override
	{
		printf("Sub2::f %d %c!\n", b_id, sub_ch);
		b_id++;
		sub_ch++;
	}
	__host__ __device__ void g() override
	{
		printf("Sub2::g %d %c!\n", b_id, sub_ch);
	}
};

__global__ void runner(Unified<Base*>* objects, int len)
{
	for (int i=0; i<len; i++)
	{
		objects[i]->f();
	}
}

int main()
{
	unique_ptr<Sub1> s1(new Sub1(1, 2));
	unique_ptr<Sub2> s2(new Sub2(3, 'd'));
	ClassMigrator<Sub1, true> sub1_migrator(s1.get(), 1);
	ClassMigrator<Sub2> sub2_migrator(s2.get(), 1);
	unique_ptr<Unified<Base*>[]> objs(new Unified<Base*>[2]);
	objs[0] = s1.get();
	objs[1] = s2.get();
	// Migrate the objects to the device
	sub1_migrator.toDevice();
	sub2_migrator.toDevice();
	hipDeviceSynchronize();
	// Run the demo to make sure everything works.
	runner<<<1,1>>>(objs.get(), 2);
	// Migrate the objects back to the host
	sub1_migrator.toHost();
	sub2_migrator.toHost();
	// Another demo to make sure that the changes come back to the host
	objs[0]->g();
	objs[1]->g();
	hipDeviceSynchronize();
	return 0;
}
